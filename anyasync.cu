/*
__any_sync,__all_syncのサンプル。しかし仕様がよくわからない。多分もう使わない(使えない)

Deprecation notice: __any, __all, and __ballot have been deprecated in CUDA 9.0 for all devices.

Removal notice: When targeting devices with compute capability 7.x or higher, __any, __all, and __ballot are no longer available and their sync variants should be used instead.

非推奨の通知：__ any、__ all、および__ballotは、すべてのデバイスのCUDA9.0で非推奨になりました。

削除通知：コンピューティング機能7.x以降のデバイスを対象とする場合、__ any、__ all、および__ballotは使用できなくなり、代わりにそれらの同期バリアントを使用する必要があります。
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void anyatest(int *A,int *B) 
{
	unsigned int tx = threadIdx.x;
	//unsigned int mask = 0xffffffff;
	unsigned int mask = 0x0000001f;
	B[tx] = __any_sync(mask, A[tx]);
}




int main() {
	int N = 128;
	int* h_A = (int*)malloc(N * sizeof(int));
	int* h_B = (int*)malloc(N * sizeof(int));
	int *d_A,*d_B;
	hipMalloc(&d_A, N * sizeof(int));
	hipMalloc(&d_B, N * sizeof(int));
	
	for(int i=0;i<N;i++)
		h_A[i]=0;
	h_A[30]=1;
	h_A[31]=1;
	h_A[32+2]=1;
	
	//HostToDevice
	hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);

	anyatest <<<1, N >>> (d_A,d_B);
	hipMemcpy(h_B, d_B, N * sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i=0;i<N;i++){
		printf("%d",h_B[i]);
		if (i%16==15)printf("\n");
	}

	hipFree(d_A);
	hipFree(d_B);
	return 0;
}
